#include "array_sum.h" 

void array_sum(int N, double *host_array, double *tot_sum){
  
  const int n_threads_blk=256; //number of threads per block 
  
  //calculate block size 
  int num_blks = (N + n_threads_blk - 1) / n_threads_blk;
  
  //allocate device array and copy host array data to device
  double *device_array;
  hipMalloc((void**)&device_array, N*sizeof(double));
  hipMemcpy(device_array, host_array, N*sizeof(double), 
      hipMemcpyHostToDevice);

  //ptr for storing partial result on device 
  double *d_sum;
  hipMalloc((void**)&d_sum, num_blks*sizeof(double));
  
  //host ptr for copying partial results from device 
  double *h_sum = (double*)malloc(num_blks*sizeof(double));
  
  //call cuda kernel to compute sum of an array
  array_sum_kernel<<<num_blks, n_threads_blk>>>(device_array, N, d_sum);

  //copy partial sum to gpu 
  hipMemcpy(h_sum, d_sum, num_blks*sizeof(double), hipMemcpyDeviceToHost);

  for(int i=0; i<num_blks; i++){
    *tot_sum+=h_sum[i];
  } 
}
