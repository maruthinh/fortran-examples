#include "hip/hip_runtime.h"
#include "array_sum.h"

__global__ void array_sum_kernel(double *a, int n, double *sum){

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  double temp=0.0;

  //calculate partial sum 
  while (tid < n){
    temp += a[tid];
    tid += blockDim.x*gridDim.x;
  }

  //store the partial sum in shared memory 
  __shared__ double part_sum[256];
  part_sum[threadIdx.x] = temp;
  __syncthreads();

  //reduce partial sum 
  int i=blockDim.x/2;
  while(i != 0 ){
    if(threadIdx.x < i) {
      part_sum[threadIdx.x] += part_sum[threadIdx.x + i];
    }
    __syncthreads();
    i /= 2;
  }

  //Write final result to sum 
  if(threadIdx.x == 0){
    sum[blockIdx.x] = part_sum[0];
  }
}

